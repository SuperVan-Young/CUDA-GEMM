#include "kernel/sgemm.cuh"

typedef cks::common::retCode_t retCode_t;

namespace cks {namespace sgemm{

void sgemmKernel_cuBLAS(int M, int N, int K, const float *h_alpha, const float *h_beta, 
                        const float *d_A, const float *d_B, float *d_C) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 
                h_alpha, d_A, M, d_B, K, h_beta, d_C, M);
    hipblasDestroy(handle);
}

}};  // namespace cks::sgemm