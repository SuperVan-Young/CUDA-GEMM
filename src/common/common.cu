#include "common/common.cuh"

namespace cks { namespace common {

retCode_t runKernel(int kernel, int version, KernelArgs* args) {
    switch (kernel) {
        case KER_SGEMM: return cks::sgemm::runKernel(version, (cks::sgemm::SgemmArgs *)args);
        default: return RC_ERROR;
    }
}

bool verifyKernel(int kernel, int version, KernelArgs *args) {
    KernelArgs *args_copy = nullptr;

    CKS_CALL(args->deepcopy(&args_copy));

    CKS_CALL(runKernel(kernel, version, args));
    CKS_CALL(runKernel(kernel, version, args_copy));

    retCode_t ret;
    if (args->equalResult(args_copy))
        ret = RC_SUCCESS;
    else
        ret = RC_ERROR;
    
    CKS_CALL(args->destroyCopy(args_copy));
    return ret;
}

float speedTestKernel(int kernel, int version, KernelArgs *args) {
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    CUDA_CALL(hipEventRecord(start));
    hipEventQuery(start);

    CKS_CALL(runKernel(kernel, version, args));

    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    float elapsed_time;
    CUDA_CALL(hipEventElapsedTime(&elapsed_time, start, stop));
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));

    return elapsed_time;
}

DataLoader *createDataLoader(int kernel) {
    DataLoader *p = nullptr;
    switch (kernel) {
        case KER_SGEMM: p = new cks::sgemm::SgemmDataLoader(); break;
        default: break;
    }
    return p;  // should delete p in the end!
}

void destroyDataLoader(DataLoader *p) {
    delete p;
    return;
}

}}  // namespace cks::common